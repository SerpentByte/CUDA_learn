#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<ctime>
#include<cstdlib>
#include<cstdio>
#include<unistd.h>
#include<chrono>
#include<omp.h>
#include<cmath>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>


using namespace std;

__global__ void sum(thrust::device_vector<float>* v1, 
	thrust::device_vector<float>* v2, float* sum, size_t n)
{
	size_t id = blockIdx.x*blockDim.x + threadIdx.x;
	sum[0] += v1[id]+v2[id];
}

float prng(int N=100)
{
	srand(getppid()*int(clock())*rand());
	float r = N*float(rand())/float(RAND_MAX);
	return r;
}

int main(int argc, char* argv[])
{
	//system("clear");

	size_t n = 2e1;
	float *sum = malloc(sizeof(float));
	float *dev_sum;

	if(argc>1)
	{
		n = (size_t)atoi(argv[1]);
	}

	thrust::host_vector<float> hv1(n,0);
	thrust::host_vector<float> hv2(n,0);
	thrust::device_vector<float> dv1(n,0);
	thrust::device_vector<float> dv2(n,0);

	hipMalloc((void**)&dev_sum, sizeof(float));

	for(size_t i=0;i<n;i++)
	{
		hv1[i] = 1;
		hv2[i] = 1;
	}

	dv1 = hv1; dv2 = hv2;
	hipMemcpy(dev_sum, &sum, sizeof(sum), hipMemcpyHostToDevice);
	sum<<<1+(n/256)>>>(dv1, dv2, dev_sum, n);
	hipMemcpy(&sum, dev_sum, sizeof(sum), hipMemcpyDeviceToHost);

	cout<<"Sum= "<<sum<<endl;

	return 0;
}