#include<iostream>
#include "hip/hip_runtime.h"

#include<ctime>
#include<cstdlib>
#include<cstdio>
#include<unistd.h>

using namespace std;

int prng(int N=100)
{
	srand(getppid()*int(clock())*rand());
	int r = int(N*float(rand())/float(RAND_MAX));
	return r;
}

__global__ void add_arrays(int *a, int *b, int n)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<n)
	{
		a[id]+=1;
	}
}

int main()
{
	system("clear");

	unsigned long long int n = 10;
	int *a = (int*)malloc(n*sizeof(int));
	int *b = (int*)malloc(n*sizeof(int));
	int *c = (int*)malloc(n*sizeof(int));

	for(unsigned long long int i=0;i<n;i++)
	{
		a[i] = prng();
		b[i] = prng();
		c[i] = a[i];
	}

	int *dev_a, *dev_b;//, *dev_c;

	if(hipMalloc((void**)&dev_a, sizeof(int)*n)==hipSuccess
		&& hipMalloc((void**)&dev_b, sizeof(int)*n)==hipSuccess)
		//&& cudaMalloc((void**)&dev_c, sizeof(int)*n)==cudaSuccess)
	{
		NULL;
	}
	else
	{
		cout<<"Failed to allocate memory on device. Exiting.";
		exit(0);
	}

	hipMemcpy(dev_a, &a, sizeof(a)*n,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, sizeof(b)*n,hipMemcpyHostToDevice);
	//cudaMemcpy(dev_c, &c, sizeof(c)*n,cudaMemcpyHostToDevice);

	add_arrays<<<1,n>>>(dev_a, dev_b, n);
	hipMemcpy(&a, dev_a, sizeof(c)*n,hipMemcpyDeviceToHost);


	for(unsigned long long int i=0;i<n;i++)
	{
		//cout<<a[i]<<'+'<<b[i]<<"="<<c[i]<<endl;
		cout<<c[i]<<' '<<a[i]<<endl;
	}

	free(a); free(b); free(c);
	hipFree(dev_a); hipFree(dev_b); //cudaFree(dev_c);

	return 0;
}