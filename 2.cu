#include<iostream>
#include "hip/hip_runtime.h"

#include<ctime>
#include<cstdlib>
#include<cstdio>
#include<unistd.h>

using namespace std;

int prng(int N=100)
{
	srand(getppid()*int(clock())*rand());
	int r = int(N*float(rand())/float(RAND_MAX));
	return r;
}

__global__ void add_arrays(int *a, int *b, int *c, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main()
{
	system("clear");

	unsigned long long int n = 10;
	int *a = (int*)malloc(n*sizeof(int));
	int *b = (int*)malloc(n*sizeof(int));
	int *c = (int*)malloc(n*sizeof(int));

	for(unsigned long long int i=0;i<n;i++)
	{
		a[i] = prng();
		b[i] = prng();
		c[i] = 0;
	}

	int *dev_a, *dev_b, *dev_c;

	if(hipMalloc((void**)&dev_a, sizeof(int)*n)==hipSuccess
		&& hipMalloc((void**)&dev_b, sizeof(int)*n)==hipSuccess
		&& hipMalloc((void**)&dev_c, sizeof(int)*n)==hipSuccess)
	{
		NULL;
	}
	else
	{
		cout<<"Failed to allocate memory on device. Exiting.";
		exit(0);
	}

	hipMemcpy(dev_a, a, sizeof(int)*n,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(int)*n,hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, sizeof(int)*n,hipMemcpyHostToDevice);

	add_arrays<<<1,n>>>(dev_a, dev_b, dev_c, n);
	hipMemcpy(c, dev_c, sizeof(int)*n,hipMemcpyDeviceToHost);


	for(unsigned long long int i=0;i<n;i++)
	{
		cout<<a[i]<<'+'<<b[i]<<"="<<c[i]<<endl;
	}

	free(a); free(b); free(c);
	hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

	return 0;
}