#include<iostream>
#include<hip/hip_runtime.h>

#include<ctime>
#include<cstdlib>
#include<cstdio>
#include<unistd.h>
#include<chrono>
#include<omp.h>

using namespace std;

int prng(int N=100)
{
	srand(getppid()*int(clock())*rand());
	int r = int(N*float(rand())/float(RAND_MAX));
	return r;
}

__global__ void add_arrays(int *a, int *b, int *c, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

void display(int *A, int n)
{
	for(size_t i=0;i<n;i++)
	{
		cout<<A[i]<<' ';
	}
	cout<<endl;
}

int main(int argc, char* argv[])
{
	system("clear");

	size_t n = 1e7;
	if(argc>1)
	{
		n = (size_t)atoi(argv[1]);
	}
	int *a = (int*)malloc(n*sizeof(int));
	int *b = (int*)malloc(n*sizeof(int));
	int *c = (int*)malloc(n*sizeof(int));

	for(size_t i=0;i<n;i++)
	{
		a[i] = prng();
		b[i] = prng();
		c[i] = 0;
	}

	int *dev_a, *dev_b, *dev_c;

	if(hipMalloc((void**)&dev_a, sizeof(int)*n)==hipSuccess
		&& hipMalloc((void**)&dev_b, sizeof(int)*n)==hipSuccess
		&& hipMalloc((void**)&dev_c, sizeof(int)*n)==hipSuccess)
	{
		NULL;
	}
	else
	{
		cout<<"Failed to allocate memory on device. Exiting.";
		exit(0);
	}

	hipMemcpy(dev_a, a, sizeof(int)*n,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(int)*n,hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, sizeof(int)*n,hipMemcpyHostToDevice);

	//cout<<"Arrays copied to device."<<endl;

	auto start = chrono::high_resolution_clock::now();
	add_arrays<<<1+(n/256),n>>>(dev_a, dev_b, dev_c, n);
	auto end = chrono::high_resolution_clock::now();

	double exec_t = chrono::duration_cast<chrono::microseconds>(end-start).count();
	hipMemcpy(c, dev_c, sizeof(int)*n,hipMemcpyDeviceToHost);

	//cout<<"Sum copied from device."<<endl;

	//display(c,n);
	cout<<"n = "<<n<<endl<<"Time taken = "<<exec_t<<" us\n";

	free(a); free(b); free(c);
	hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);



	return 0;
}