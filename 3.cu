#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<ctime>
#include<cstdlib>
#include<cstdio>
#include<unistd.h>
#include<chrono>
#include<omp.h>
#include<cmath>

using namespace std;

struct xyz
{
	double x, y ,z;
	xyz()
	{
		x=0; y=0; z=0;
	}

	xyz(double a, double b, double c)
	{
		x=a; y=b; z=c;
	}
};

float prng(int N=100)
{
	srand(getppid()*int(clock())*rand());
	float r = N*float(rand())/float(RAND_MAX);
	return r;
}

__global__ void nearest(xyz* points, int* nn, int n)
{
	if(n<=1)
	{
		return;
	}
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	xyz priv_point;

	if(id<n)
	{
		priv_point = points[id];
		double d = 1e37, distance=0;;
		for(size_t i=0;i<n;i++)
		{
			if(i==id) continue;
			distance = pow((priv_point.x - points[i].x),2);
			distance += pow((priv_point.y - points[i].y),2);
			distance += pow((priv_point.z - points[i].z),2);
			//distance = sqrt(distance);
			if(distance<d)
			{
				nn[id] = i+1;
				d = distance; 
			}
		}
	}

}


int main(int argc, char* argv[])
{
	//system("clear");

	size_t n = 2e1;
	if(argc>1)
	{
		n = (size_t)atoi(argv[1]);
	}

	xyz* points = (xyz*)(malloc(sizeof(xyz)*n));
	int* nn = (int*)(malloc(sizeof(int)*n));

	xyz* dev_points;
	int* dev_nn;

	for(size_t i=0;i<n;i++)
	{
		points[i].x = prng();
		points[i].y = prng();
		points[i].z = prng();
		nn[i] = 0;
	}

	hipMalloc((void**)&dev_points, sizeof(xyz)*n);
	hipMalloc((void**)&dev_nn, sizeof(int)*n);

	hipMemcpy(dev_points, points, sizeof(xyz)*n, hipMemcpyHostToDevice);
	hipMemcpy(dev_nn, nn, sizeof(int)*n, hipMemcpyHostToDevice);

	auto start = chrono::high_resolution_clock::now();
	nearest<<<1+(n/256),n>>>(points, nn, n);
	auto end = chrono::high_resolution_clock::now();

	double exec_t = chrono::duration_cast<chrono::nanoseconds>(end-start).count();
	hipMemcpy(nn, dev_nn, sizeof(int)*n, hipMemcpyDeviceToHost);

	cout<<"Time taken = "<<exec_t<<" ns\n";

	free(points); free(nn);
	hipFree(dev_points); hipFree(dev_nn);

	return 0;
}