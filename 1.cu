#include<iostream>
#include "hip/hip_runtime.h"

#include<ctime>
#include<cstdlib>
#include<cstdio>
#include<unistd.h>

using namespace std;

int prng(int N=100)
{
	srand(getppid()*int(clock())*rand());
	int r = int(N*float(rand())/float(RAND_MAX));
	return r;
}

__global__ void add(int *a, int *b)
{
	a[0] += b[0];
}



int main()
{

	int a = 3, b = 5; 
	cout<<"Enter value of a: "; cin>>a;
	cout<<"Enter value of b: "; cin>>b;
	int *dev_a, *dev_b;

	hipMalloc((void**)&dev_a, sizeof(int));
	hipMalloc((void**)&dev_b, sizeof(int));
	hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, sizeof(int), hipMemcpyHostToDevice);

	add<<<1,1>>>(dev_a, dev_b);
	hipMemcpy(&a, dev_a, sizeof(int), hipMemcpyDeviceToHost);

	cout<<"a+b = "<<a<<endl;
	hipFree(dev_a);
	hipFree(dev_b);


	return 0;
}